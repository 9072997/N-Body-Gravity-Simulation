/*  
	N-Body Gravity Simulation
	Copyright (C) 2014 Jon Penn

	This program is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
const int kbodyCount = 1;
const int ktickCount = 1024*1024*1024;

int signAtBit(unsigned int bits, int position) {
	return -((bits&(1<<position))>>position)*2+1;
}

__global__ void gravify(float *bodyMass, float *bodyXposIn,  float *bodyYposIn,  float *bodyXvel,  float *bodyYvel,
										 float *bodyXposOut, float *bodyYposOut, int bodyCount) {
	int numBody;
	numBody = blockIdx.x*blockDim.x + threadIdx.x;
	//if(numBody >= bodyCount) return; // in case we have left over threads
	for(int numInfl = 0; numInfl < bodyCount; numInfl++) {
		float scale, dx, dy;
		if(bodyXposIn[numInfl]==bodyXposIn[numBody] && bodyYposIn[numInfl]==bodyYposIn[numBody]) continue; // 2 points at the same position (or we measure ourself)
		dx = bodyXposIn[numInfl] - bodyXposIn[numBody];
		dy = bodyYposIn[numInfl] - bodyYposIn[numBody];
		scale = bodyMass[numInfl] * pow(pow(dx,2) + pow(dy,2), -3/2); // initialy multiply body mass by g
		bodyXvel[numBody] += scale * dx;
		bodyYvel[numBody] += scale * dy;
	}
	bodyXposOut[numBody] = bodyXposIn[numBody] + bodyXvel[numBody];;
	bodyYposOut[numBody] = bodyYposIn[numBody] + bodyYvel[numBody];
}

int main() {
	FILE *file;
	float *bodyMass,    *bodyXpos,    *bodyYpos,   *bodyXvel,   *bodyYvel;
	float *d_bodyMass, **d_bodyXpos, **d_bodyYpos, *d_bodyXvel, *d_bodyYvel;
	
	bodyMass = (float *)malloc(kbodyCount*1024*sizeof(float));
	bodyXpos = (float *)malloc(kbodyCount*1024*sizeof(float));
	bodyYpos = (float *)malloc(kbodyCount*1024*sizeof(float));
	bodyXvel = (float *)malloc(kbodyCount*1024*sizeof(float));
	bodyYvel = (float *)malloc(kbodyCount*1024*sizeof(float));
	
	d_bodyXpos = (float **)malloc(2*sizeof(float *));
	d_bodyYpos = (float **)malloc(2*sizeof(float *));
	
	hipMalloc(&d_bodyMass,    kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyXpos[0], kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyXpos[1], kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyYpos[0], kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyYpos[1], kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyXvel,    kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMalloc(&d_bodyYvel,    kbodyCount*1024*sizeof(float)) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	
	for(int numBody = 0; numBody < kbodyCount*1024/16; numBody++) { // each iliteration creates 16 bodys
		float mass, xpos, ypos, xvel, yvel;
		mass = abs((numBody+8156897)*49459879%500+1);
		xpos =     (numBody+5867952)*89526249%654654;
		ypos =     (numBody+7352405)*68724646%687984;
		xvel =     (numBody+8987354)*25897895%795;
		yvel =     (numBody+9444555)*16871232%826;
		
		for(int bits = 0; bits < 16; bits++) { // 16 bodys
			bodyMass[numBody*16+bits] = mass;
			bodyXpos[numBody*16+bits] = signAtBit(bits,0)*xpos;
			bodyYpos[numBody*16+bits] = signAtBit(bits,1)*ypos;
			bodyXvel[numBody*16+bits] = signAtBit(bits,2)*xvel;
			bodyYvel[numBody*16+bits] = signAtBit(bits,3)*yvel;
		}
	}
	
	file = fopen("masses.csv", "w");
	for(int numBody = 0; numBody < kbodyCount*1024; numBody++) {
		fprintf(file, "%f\n", bodyMass[numBody]);
	}
	fclose(file);
	
	hipMemcpy(d_bodyMass,    bodyMass, kbodyCount*1024*sizeof(float), hipMemcpyHostToDevice) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_bodyXpos[0], bodyXpos, kbodyCount*1024*sizeof(float), hipMemcpyHostToDevice) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_bodyYpos[0], bodyYpos, kbodyCount*1024*sizeof(float), hipMemcpyHostToDevice) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_bodyXvel,    bodyXvel, kbodyCount*1024*sizeof(float), hipMemcpyHostToDevice) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_bodyYvel,    bodyYvel, kbodyCount*1024*sizeof(float), hipMemcpyHostToDevice) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
	
	
	for(int numKtick = 0; numKtick < ktickCount; numKtick++) {
		char filename[60]; // 20 should be ok, but why not
		fprintf(stderr, "ktick: %d\n", numKtick);
		for(int numTick = 0; numTick < 512; numTick++) { // each loop is accuialy 2 ticks
			gravify<<< kbodyCount, 1024 >>>(d_bodyMass, d_bodyXpos[0], d_bodyYpos[0], d_bodyXvel, d_bodyYvel,
													   d_bodyXpos[1], d_bodyYpos[1], kbodyCount*1024);								
			// reverse, reverse!
			gravify<<< kbodyCount, 1024 >>>(d_bodyMass, d_bodyXpos[1], d_bodyYpos[1], d_bodyXvel, d_bodyYvel,
													   d_bodyXpos[0], d_bodyYpos[0], kbodyCount*1024);
		}
		// note here we assume we have made an even number of ilitrations
		//cudaMemcpy(bodyMass, d_bodyMass,  kbodyCount*1024*sizeof(float), cudaMemcpyDeviceToHost);
		hipMemcpy(bodyXpos, d_bodyXpos[0], kbodyCount*1024*sizeof(float), hipMemcpyDeviceToHost) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
		hipMemcpy(bodyYpos, d_bodyYpos[0], kbodyCount*1024*sizeof(float), hipMemcpyDeviceToHost) ==hipSuccess||printf("(FAIL 1 : %s)", hipGetErrorString(hipGetLastError()));
		//cudaMemcpy(bodyXvel, d_bodyXvel,  kbodyCount*1024*sizeof(float), cudaMemcpyDeviceToHost);
		//cudaMemcpy(bodyYvel, d_bodyYvel,  kbodyCount*1024*sizeof(float), cudaMemcpyDeviceToHost);
		
		sprintf(filename, "ktick%010d.csv", numKtick);
		file = fopen(filename, "w");
		for(int numBody = 0; numBody < kbodyCount*1024; numBody++) {
			fprintf(file, "%f\t%f\n", bodyXpos[numBody], bodyYpos[numBody]);
		}
		fclose(file);
	}
	
	free(bodyMass);
	free(bodyXpos);
	free(bodyYpos);
	free(bodyXvel);
	free(bodyYvel);
	
	hipFree(d_bodyMass);
	hipFree(d_bodyXpos[0]);
	hipFree(d_bodyXpos[1]);
	hipFree(d_bodyYpos[1]);
	hipFree(d_bodyYpos[0]);
	hipFree(d_bodyXvel);
	hipFree(d_bodyYvel);
	
	free(d_bodyXpos);
	free(d_bodyYpos);
	
	return 0;
}
